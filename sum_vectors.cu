#include <iostream>
#include <random>
#include "hip/hip_runtime.h"
#include <stdlib.h>

__global__ void sumVectorCUDA(int *a, int *b, int count){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < count){
        a[id] += b[id];
    }
}

int main(int argc,char **argv){
    
    std::random_device rd;
    std::mt19937 gen(0);
    std::uniform_real_distribution<double> un_dis(0,1000);
    
    int count = 10; //Size of each array
    int *h_a = nullptr;
    int *h_b = nullptr;
    h_a = new int[count]; //Memory 
    h_b = new int[count]; //Host version

    for (int ii =0; ii < count; ++ii){
        h_a[ii] = un_dis(gen);
        h_b[ii] = un_dis(gen);
    }
    for (int pp =0; pp < 5; ++pp){
        std::cout << h_a[pp] << "\t" << h_b[pp] << std::endl;
    }

    //Here we go
    
    int *d_a, *d_b; //Device
    //Memory allocation on device
    if(hipMalloc(&d_a, sizeof(int)*count) != hipSuccess){
        std::cout << "error" << std::endl;
        return 0;
    }
    if(hipMalloc(&d_b, sizeof(int)*count) != hipSuccess){
        std::cout << "error" << std::endl;
        return 0;
    }
    //Copy array values
    hipMemcpy(d_a, h_a, sizeof(int)*count,hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*count,hipMemcpyHostToDevice);

    sumVectorCUDA<<<5,2>>>(d_a,d_b,count);

    hipMemcpy(h_a, d_a, sizeof(int)*count,hipMemcpyDeviceToHost);

    for (int ff =0; ff < 5; ++ff){
        std::cout << h_a[ff] << std::endl;
    }
    
    hipFree(d_a);
    hipFree(d_b);
    
    delete[] h_a;
    delete[] h_b;
    
    return 0;
    
}
