#include <iostream>
#include "hip/hip_runtime.h"

__global__ void AddIntsCUDA(int*a, int *b)
{
    for (int ii = 0; ii < 10000005; ii++){
        a[0] += b[0];
    }
}

int main(int argc, char **argv){
    
    int a = 0, b = 1;
    int *d_a, *d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA<<<1,1>>>(d_a,d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "The answer is " << a << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    
    return 0;
}
